#include "hip/hip_runtime.h"
//
//  Ray.cpp
//  RayTracer
//
//  Created by Bo Miller on 1/2/19.
//  Copyright © 2019 Bo Miller. All rights reserved.
//LDFLAGS= -L./glm/glm -glm
#include "glm/glm/glm.hpp"
#include "glm/glm/gtx/io.hpp"
#include <iostream>
#include "Ray.hpp"
#include <vector>
#include <atomic>
#include <mutex>
#include <math.h>
#define PI 3.14159265359
#include <pthread.h>
#include <chrono>
#include "bvh.hpp"
#include <random>
#include <queue>
#include "isect.hpp"

float RAY_EPSILON = 0.000000001;
int antialiasing = 0;
int numBounces = 1;
int numThreads = 4;
int SampPerPix = 4;
Node* root;
int totalRaysInSystem = 0;

__global__ void GeneratePrimaryRays(Ray* rays, int n, glm::vec3 L, glm::vec3 u, glm::vec3 v, glm::vec3 cameraPosition)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < n; i+=stride)
    {
	    int ipix = i%720;
	    int jpix = i/720;
        glm::vec3 pix = (L+u*float(ipix)+v*float(jpix));

        float xoffset = 0;
        float yoffset = 0;
        glm::vec3 sample = glm::normalize(glm::vec3(pix[0]+xoffset,pix[1]+yoffset,pix[2])-cameraPosition);
        rays[i].raytype = 0;
        rays[i].timesbounced = 0;
        rays[i].position = cameraPosition;
        rays[i].direction = sample;
        rays[i].i = ipix;
        rays[i].j = jpix;
        rays[i].color = glm::vec3(0,0,0);
    }
}

__device__ bool boundingBoxIntersection(glm::vec3 position, glm::vec3 direction, Node* node)
{
    float tmin = (node->minX-position[0])/direction[0];
    float tmax = (node->maxX-position[0])/direction[0];
    
    if(tmin>tmax)
    {
        float temp = tmin;
        tmin = tmax;
        tmax = temp;
    }
    float tymin = (node->minY-position[1])/direction[1];
    float tymax = (node->maxY-position[1])/direction[1];
    
    if(tymin>tymax)
    {
        float temp = tymin;
        tymin = tymax;
        tymax = temp;
    }
    
    if((tmin > tymax) || (tymin > tmax))
        return false;
    
    if (tymin > tmin)
        tmin = tymin;
    
    if (tymax < tmax)
        tmax = tymax;
    
    float tzmin = (node->minZ-position[2])/direction[2];
    float tzmax = (node->maxZ-position[2])/direction[2];
    
    if (tzmin > tzmax)
    {
        float temp = tzmin;
        tzmin = tzmax;
        tzmax = temp;
    }
    
    if ((tmin > tzmax) || (tzmin > tmax))
        return false;
    
    if (tzmin > tmin)
        tmin = tzmin;
    
    if (tzmax < tmax)
        tmax = tzmax;
    
    return true;
}

__device__ void bvhTraverse(glm::vec3& position, glm::vec3& direction, Node* currentNode,bool& intersect,float& minT, SceneObject& intersectObj, glm::vec3& minTnormal, glm::vec3& minTintersection)
{
    float RAY_EPSILON = 0.000000001;
    if(currentNode->isleaf)
    {
        if(boundingBoxIntersection(position, direction, currentNode))
        {
            for(int i = 0; i<currentNode->numObjs;i++)
            {
                SceneObject s = currentNode->objs[i];
                if(s.sphere)
                {
                        
                    glm::vec3 normal;
                    glm::vec3 intersection;
                    bool sphereintersected = false;;
                        
                    float iTime;
                    float a = glm::dot(direction, direction);
                    float b = 2 * glm::dot(direction,position-s.center);
                    float c = glm::dot(s.center,s.center) + glm::dot(position,position) + (-2 * glm::dot(s.center,position)) - pow(s.radius,2);
                    
                    float discriminant = b*b - 4*a*c;
                    
                    if(discriminant > 0.0+RAY_EPSILON)
                    {
                    
                        float t = (-b - sqrt(discriminant))/(2*a);
                    
                        float t2 = (-b + sqrt(discriminant))/(2*a);
                    
                    
                        if(t2>RAY_EPSILON)
                        {
                            //we know we have some intersection
                    
                            if( t > RAY_EPSILON )
                            {
                                iTime = t;
                            }
                            else
                            {
                                iTime = t2;
                            }
                                
                            intersection = position+t*direction;
                            normal = glm::normalize((intersection-s.center)/s.radius);
                            sphereintersected = true;
                        }
                    }

                    if(sphereintersected)
                    {
                        if(iTime<minT)
                        {
                            minTnormal = normal;
                            minTintersection = intersection;
                            intersectObj = s;
                            minT = iTime;
                            intersect = true;
                        }
                    }
                }   
            }
        }
    }
    else
    {
        if(boundingBoxIntersection(position, direction, currentNode->left))
            bvhTraverse(position, direction,currentNode->left,intersect,minT,intersectObj,minTnormal,minTintersection);
        if(boundingBoxIntersection(position, direction, currentNode->right))
            bvhTraverse(position, direction,currentNode->right,intersect,minT,intersectObj,minTnormal,minTintersection);
    }
}

__device__ bool wallIntersection(Isect& ipoint, Ray& r, Ray& reflect)
{
    glm::vec3 up = glm::vec3(0,1,0);
    float denom = glm::dot(up,r.direction);
    if(fabsf(denom) > .0001f)
    {
        float t = glm::dot((glm::vec3(0,-2,0)-r.position),up)/denom;
        if(t >= 0.0-.0001f)
        {
            glm::vec3 intersect = r.position+t*r.direction;
            
            SceneObject wall;
            wall.ambient = glm::vec3(1.0, 0.2, 0.2);
            wall.diffuse = glm::vec3(1.0, 0.2, 0.2);
            wall.specular = glm::vec3(0.0,0.0,0.0);
            wall.shininess = 2;
            wall.reflective = glm::vec3(0.0,0.0,0.0);
            
            if(intersect[2]>-15 && intersect[2]<13 && intersect[0]>-6 && intersect[0] < 6)
            {
                if(r.raytype == 0)
                {
                    ipoint.color = 0.2f * wall.ambient;
                    reflect.surfaceReflectiveCoef = wall.reflective;
                    ipoint.reflectionCoef = glm::vec3(1.0f,1.0f,1.0f);
                }
                else if(r.raytype == 1){
                    ipoint.reflectionCoef = r.surfaceReflectiveCoef;
                    reflect.surfaceReflectiveCoef = r.surfaceReflectiveCoef*wall.reflective;
                }
                ipoint.normal = up;
                ipoint.isectPoint = intersect;
                ipoint.incidentDirection = glm::normalize(glm::reflect(r.direction, up));
                ipoint.diffuse = wall.diffuse;
                ipoint.ambient = wall.ambient;
                ipoint.specular = wall.specular;
                ipoint.shininess = wall.shininess;
                ipoint.reflective = wall.reflective;
                return true;
            }
        }
    }
    
    
    //left wall
    up = glm::vec3(1,0,0);
    denom = glm::dot(up,r.direction);
    if(abs(denom) > .0001f)
    {
        float t = glm::dot((glm::vec3(-6,0,0)-r.position),up)/denom;
        if(t >= 0.0-.0001f)
        {
            glm::vec3 intersect = r.position+t*r.direction;
            
            SceneObject wall;
            wall.ambient = glm::vec3(0.2, 0.2, 1.0);
            wall.diffuse = glm::vec3(0.2, 0.2, 1.0);
            wall.specular = glm::vec3(0.0,0.0,0.0);
            wall.reflective = glm::vec3(0.0,0.0,0.0);
            wall.shininess = 2;
            
            if(intersect[2]>-15 && intersect[2] < 13 && intersect[1] < 9 && intersect[1]>-2)
            {
                if(r.raytype == 0)
                {
                    ipoint.color = 0.2f * wall.ambient;
                    reflect.surfaceReflectiveCoef = wall.reflective;
                    ipoint.reflectionCoef = glm::vec3(1.0f,1.0f,1.0f);
                }
                else if(r.raytype == 1){
                    ipoint.reflectionCoef = r.surfaceReflectiveCoef;
                    reflect.surfaceReflectiveCoef = r.surfaceReflectiveCoef*wall.reflective;
                }
                ipoint.normal = up;
                ipoint.isectPoint = intersect;
                ipoint.incidentDirection = glm::normalize(glm::reflect(r.direction, up));
                ipoint.diffuse = wall.diffuse;
                ipoint.ambient = wall.ambient;
                ipoint.specular = wall.specular;
                ipoint.shininess = wall.shininess;
                ipoint.reflective = wall.reflective;
                return true;
            }
        }
    }
    
    //front wall, green wall in front of camera
    up = glm::vec3(0,0,1);
    denom = glm::dot(up,r.direction);
    if(abs(denom) > .0001f)
    {
        float t = glm::dot((glm::vec3(0,0,-15)-r.position),up)/denom;
        if(t >= 0.0-.0001f)
        {
            glm::vec3 intersect = r.position+t*r.direction;
            
            SceneObject wall;
            wall.ambient = glm::vec3(0.0, 1.0, 0.0);
            wall.diffuse = glm::vec3(0.0, 1.0, 0.0);
            wall.specular = glm::vec3(0.0,0.0,0.0);
            wall.shininess = 2;
            wall.reflective = glm::vec3(0.0,0.0,0.0);
            
            if(intersect[0] < 6 && intersect[0] > -6 && intersect[1] < 9 && intersect[1] > -2 )
            {
                if(r.raytype == 0)
                {
                    ipoint.color = 0.2f * wall.ambient;
                    reflect.surfaceReflectiveCoef = wall.reflective;
                    ipoint.reflectionCoef = glm::vec3(1.0f,1.0f,1.0f);
                }
                else if(r.raytype == 1){
                    ipoint.reflectionCoef = r.surfaceReflectiveCoef;
                    reflect.surfaceReflectiveCoef = r.surfaceReflectiveCoef*wall.reflective;
                }
                ipoint.normal = up;
                ipoint.isectPoint = intersect;
                ipoint.incidentDirection = glm::normalize(glm::reflect(r.direction, up));
                ipoint.diffuse = wall.diffuse;
                ipoint.ambient = wall.ambient;
                ipoint.specular = wall.specular;
                ipoint.shininess = wall.shininess;
                ipoint.reflective = wall.reflective;
                return true;
            }
        }
    }
    //back wall, yellow wall behind camera
    up = glm::vec3(0,0,-1);
    denom = glm::dot(up,r.direction);
    if(abs(denom) > .0001f)
    {
        float t = glm::dot((glm::vec3(0,0,13)-r.position),up)/denom;
        if(t >= 0.0-.0001f)
        {
            glm::vec3 intersect = r.position+t*r.direction;
            
            SceneObject wall;
            wall.ambient = glm::vec3(1.0, 1.0, 0.0);
            wall.diffuse = glm::vec3(1.0, 1.0, 0.0);
            wall.specular = glm::vec3(0.0,0.0,0.0);
            wall.shininess = 2;
            wall.reflective = glm::vec3(0.0,0.0,0.0);
            
            if(intersect[0] < 6 && intersect[0] > -6 && intersect[1] < 9 && intersect[1] > -2 )
            {
                if(r.raytype == 0)
                {
                    ipoint.color = 0.2f * wall.ambient;
                    reflect.surfaceReflectiveCoef = wall.reflective;
                    ipoint.reflectionCoef = glm::vec3(1.0f,1.0f,1.0f);
                }
                else if(r.raytype == 1){
                    ipoint.reflectionCoef = r.surfaceReflectiveCoef;
                    reflect.surfaceReflectiveCoef = r.surfaceReflectiveCoef*wall.reflective;
                }
                ipoint.normal = up;
                ipoint.isectPoint = intersect;
                ipoint.incidentDirection = glm::normalize(glm::reflect(r.direction, up));
                ipoint.diffuse = wall.diffuse;
                ipoint.ambient = wall.ambient;
                ipoint.specular = wall.specular;
                ipoint.shininess = wall.shininess;
                ipoint.reflective = wall.reflective;
                
                return true;
            }
        }
    }
    
    //right wall
    up = glm::vec3(-1,0,0);
    denom = glm::dot(up,r.direction);
    if(abs(denom) > .0001f)
    {
        float t = glm::dot((glm::vec3(6,0,0)-r.position),up)/denom;
        if(t >= 0.0-.0001f)
        {
            glm::vec3 intersect = r.position+t*r.direction;
            
            SceneObject wall;
            wall.ambient = glm::vec3(0.2, 0.2, 1.0);
            wall.diffuse = glm::vec3(0.2, 0.2, 1.0);
            wall.specular = glm::vec3(0.0,0.0,0.0);
            wall.shininess = 2;
            wall.reflective = glm::vec3(0.0,0.0,0.0);
            
            if(intersect[2]>-15 && intersect[2] < 13 && intersect[1] < 9 && intersect[1]>-2)
            {
                if(r.raytype == 0)
                {
                    ipoint.color = 0.2f * wall.ambient;
                    reflect.surfaceReflectiveCoef = wall.reflective;
                    ipoint.reflectionCoef = glm::vec3(1.0f,1.0f,1.0f);
                }
                else if(r.raytype == 1){
                    ipoint.reflectionCoef = r.surfaceReflectiveCoef;
                    reflect.surfaceReflectiveCoef = r.surfaceReflectiveCoef*wall.reflective;
                }
                ipoint.normal = up;
                ipoint.isectPoint = intersect;
                ipoint.incidentDirection = glm::normalize(glm::reflect(r.direction, up));
                ipoint.diffuse = wall.diffuse;
                ipoint.ambient = wall.ambient;
                ipoint.specular = wall.specular;
                ipoint.shininess = wall.shininess;
                ipoint.reflective = wall.reflective;
                return true;
            }
        }
    }
    
    //ceiling
    up = glm::vec3(0,-1,0);
    denom = glm::dot(up,r.direction);
    if(abs(denom) > .0001f)
    {
        float t = glm::dot((glm::vec3(0,9,0)-r.position), up)/denom;
        if(t >= 0.0-.0001f)
        {
            glm::vec3 intersect = r.position+t*r.direction;
            
            SceneObject wall;
            wall.ambient = glm::vec3(.5, .5, .5);
            wall.diffuse = glm::vec3(.9, .9, .9);
            wall.specular = glm::vec3(0.0,0.0,0.0);
            wall.shininess = 2;
            wall.reflective = glm::vec3(0.0,0.0,0.0);
            
            if(intersect[2]>-15 && intersect[2]<13 && intersect[0]>-6 && intersect[0] < 6)
            {
                if(r.raytype == 0)
                {
                    ipoint.color = 0.2f * wall.ambient;
                    reflect.surfaceReflectiveCoef = wall.reflective;
                    ipoint.reflectionCoef = glm::vec3(1.0f,1.0f,1.0f);
                }
                else if(r.raytype == 1){
                    ipoint.reflectionCoef = r.surfaceReflectiveCoef;
                    reflect.surfaceReflectiveCoef = r.surfaceReflectiveCoef*wall.reflective;
                }
                
                ipoint.normal = up;
                ipoint.isectPoint = intersect;
                ipoint.incidentDirection = glm::normalize(glm::reflect(r.direction, up));
                ipoint.diffuse = wall.diffuse;
                ipoint.ambient = wall.ambient;
                ipoint.specular = wall.specular;
                ipoint.shininess = wall.shininess;
                ipoint.reflective = wall.reflective;
                return true;
            }
        }
    }
    r.color = glm::vec3(0,0,0);
    return false;
}

__global__ void RayIntersection(Ray* rays, int n, Ray* reflectedRays, Node* bvhhead, Isect* isectPoints, int* nw, int* ne, int* sw, int* se)
{
    float RAY_EPSILON = 0.000000001;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < n; i+=stride)
    {
        glm::vec3 direction = rays[i].direction;
        glm::vec3 position = rays[i].position;

        reflectedRays[i].i = rays[i].i;
        reflectedRays[i].j = rays[i].j;
        reflectedRays[i].color = rays[i].color;
        reflectedRays[i].raytype = 1;
        reflectedRays[i].timesbounced = rays[i].timesbounced;
        
        float minT = 1000000000;
        SceneObject intersectObj;
        glm::vec3 minTnormal;
        glm::vec3 minTintersection;
        bool intersect = false;
        bvhTraverse(position,direction,bvhhead,intersect,minT,intersectObj,minTnormal,minTintersection);
        if(intersect)
        {
            isectPoints[i].color = glm::vec3(0,0,0);
            if(rays[i].raytype == 0)
            {
                reflectedRays[i].surfaceReflectiveCoef = intersectObj.reflective;
                isectPoints[i].reflectionCoef = glm::vec3(1.0f,1.0f,1.0f);
                isectPoints[i].color = 0.2f * intersectObj.ambient;
            }
            else if(rays[i].raytype == 1){
                isectPoints[i].reflectionCoef = rays[i].surfaceReflectiveCoef;
                reflectedRays[i].surfaceReflectiveCoef = rays[i].surfaceReflectiveCoef*intersectObj.reflective;
            }
            
            minTintersection = minTintersection+minTnormal*RAY_EPSILON;
            reflectedRays[i].position = minTintersection;
            reflectedRays[i].direction = glm::normalize(glm::reflect(rays[i].direction, minTnormal));
            reflectedRays[i].validRay = true;
            
             
            isectPoints[i].isected = true; 
            isectPoints[i].isectPoint = minTintersection;
            isectPoints[i].incidentDirection = rays[i].direction;
            isectPoints[i].normal = minTnormal;
            isectPoints[i].i = rays[i].i;
            isectPoints[i].j = rays[i].j;
            
            isectPoints[i].diffuse = intersectObj.diffuse;
            isectPoints[i].ambient = intersectObj.ambient;
            isectPoints[i].shininess = intersectObj.shininess;
            isectPoints[i].specular = intersectObj.specular;
            isectPoints[i].reflective = intersectObj.reflective;

            
        }
        else
        {
            Isect point;
            wallIntersection(point,rays[i],reflectedRays[i]);
            isectPoints[i] = point;

            reflectedRays[i].position = point.isectPoint;
            reflectedRays[i].direction = glm::normalize(glm::reflect(rays[i].direction, point.normal));
            
            isectPoints[i].i = rays[i].i;
            isectPoints[i].j = rays[i].j;

        }
        if(reflectedRays[i].direction[0] <= 0.0f && reflectedRays[i].direction[1] >= 0.0f)
        {
            atomicAdd(nw,1);
        }
        else if(reflectedRays[i].direction[0] >= 0.0f && reflectedRays[i].direction[1] >= 0.0f)
        {
            atomicAdd(ne,1);
        }
        else if(reflectedRays[i].direction[0] <= 0.0f && reflectedRays[i].direction[1] <= 0.0f)
        {
            atomicAdd(sw,1);
        }
        else if(reflectedRays[i].direction[0] >= 0.0f && reflectedRays[i].direction[1] <= 0.0f)
        {
            atomicAdd(se,1);
        }
    
    }
}

__global__ void Shade(Isect* isectPoints, int n, Light* lights, int numlights, Node* bvhhead)
{

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < n; i+=stride)
    {
        glm::vec3 color;
        glm::vec3 direction = glm::normalize(isectPoints[i].incidentDirection);
        glm::vec3 intersection = isectPoints[i].isectPoint;
        glm::vec3 normal = isectPoints[i].normal;
        float distance;
        glm::vec3 toLight;
        glm::vec3 reflectFromLight;
        for(int j =0; j<numlights; j++)
        {
            Light l = lights[j];
            /*if(l.area)
            {
                std::random_device rd;  //Will be used to obtain a seed for the random number engine
                std::mt19937 gen(rd()); //Standard mersenne_twister_engine seeded with rd()
                std::uniform_real_distribution<> dis(0.0, l.radius);
                std::uniform_real_distribution<> dis2(0.0, 2*PI);
                glm::vec3 avgcolor;
                int lightSamples = 30;
                for(int i = 0 ;i<lightSamples;i++)
                {
                    float radius = dis(gen);
                    float theta = dis2(gen);
                    
                    float x = radius * cos(theta);
                    float z = radius * sin(theta);
                    l.position[0] += x;
                    l.position[2] += z;
                    toLight = glm::normalize(l.position-intersection);
                    reflectFromLight = -toLight;
                    glm::vec3 dummyC;
                    distance = 1.0f;
                    float t;
                    Ray lightRay;
                    lightRay.position = intersection;
                    lightRay.direction = toLight;
                    lightRay.color = glm::vec3(0,0,0);
                    lightRay.raytype = 2;
                    Ray dummyRay;
                    if(intersectObjects(lightRay, scene, lights, 0, t, dummyRay))
                    {
                        glm::vec3 ipoint = intersection+t*toLight;
                        float dtoLight = sqrt(pow(intersection[0]-l.position[0],2)+pow(intersection[1]-l.position[1],2)+pow(intersection[2]-l.position[2],2));
                        float dtoLightIntersection = sqrt(pow(ipoint[0]-intersection[0],2)+pow(ipoint[1]-intersection[1],2)+pow(ipoint[2]-intersection[2],2));
                        if(dtoLight>dtoLightIntersection)
                            distance = distance * 0;
                    }
                    
                    avgcolor += distance * l.color * ( .6f * s.diffuse * glm::max(glm::dot(toLight,normal),0.0f) + .2f * s.specular * glm::pow(glm::dot(glm::reflect(reflectFromLight, normal), -r.direction),s.shininess));
                }
                color += avgcolor/(float)lightSamples;
                
                
            }*/
            if(l.point)
            {
                float d = sqrt(pow(intersection[0]-l.position[0],2)+pow(intersection[1]-l.position[1],2)+pow(intersection[2]-l.position[2],2));
                distance = 1.0f/(l.constantTerm + l.linearTerm * d + l.quadraticTerm * pow(d,2));
                
                if(distance>1.5)
                    distance = .5;
                toLight = glm::normalize(l.position-intersection);
                reflectFromLight = -toLight;
                
                Ray lightRay;
                lightRay.position = intersection;
                lightRay.direction = toLight;
                lightRay.color = glm::vec3(0,0,0);
                lightRay.raytype = 2;

                float minT = 1000000000;
                SceneObject intersectObj;
                glm::vec3 minTnormal;
                glm::vec3 minTintersection;
                bool intersect = false;
                bvhTraverse(lightRay.position,lightRay.direction,bvhhead,intersect,minT,intersectObj,minTnormal,minTintersection);
                
                if(intersect)
                {
                    glm::vec3 ipoint = intersection+minT*toLight;
                    float dtoLight = sqrt(pow(intersection[0]-l.position[0],2)+pow(intersection[1]-l.position[1],2)+pow(intersection[2]-l.position[2],2));
                    float dtoLightIntersection = sqrt(pow(ipoint[0]-intersection[0],2)+pow(ipoint[1]-intersection[1],2)+pow(ipoint[2]-intersection[2],2));
                    if(dtoLight>dtoLightIntersection)
                        distance = distance * 0;
                }
                color += distance * l.color * ( .6f * isectPoints[i].diffuse * glm::max(glm::dot(toLight,normal),0.0f) + .2f * isectPoints[i].specular * glm::pow(glm::dot(glm::reflect(reflectFromLight, normal), -direction),isectPoints[i].shininess));
            }
            else
            {
                distance = 1.0f;
                toLight = -glm::normalize(l.direction);
                reflectFromLight = glm::normalize(l.direction);
                
                Ray lightRay;
                lightRay.position = intersection;
                lightRay.direction = toLight;
                lightRay.color = glm::vec3(0,0,0);
                lightRay.raytype = 2;
                

                float minT = 1000000000;
                SceneObject intersectObj;
                glm::vec3 minTnormal;
                glm::vec3 minTintersection;
                bool intersect = false;

                bvhTraverse(lightRay.position,lightRay.direction,bvhhead,intersect,minT,intersectObj,minTnormal,minTintersection);

                if(intersect)
                {
                    distance = distance * 0;
                }
                color += distance * l.color * ( .6f * isectPoints[i].diffuse * glm::max(glm::dot(toLight,normal),0.0f) + .2f * isectPoints[i].specular * glm::pow(glm::dot(glm::reflect(reflectFromLight, normal), -direction),isectPoints[i].shininess));
            }
        }
        isectPoints[i].color += isectPoints[i].reflectionCoef * color;
    }
}

void startRayTracing(float width, float height, float (&pixelcolorBuffer)[360][720][3],glm::vec3 cameraPosition, glm::vec3 cameraDirection, std::vector<SceneObject>& scene, std::vector<Light>& lights, Node* rootnode)
{

    
    Light* scenelights;
    int numlights = (int)lights.size();
    hipMallocManaged(&scenelights,numlights*sizeof(Light));
    for(int i = 0; i<numlights; i++)
    {
        scenelights[i] = lights[i];
    }

    totalRaysInSystem = width*height;
    

    root = (Node *)malloc(sizeof(Node));
    root = rootnode;
    //for primary ray calcuations
    glm::vec3 n = glm::normalize(cameraPosition-cameraDirection);
    glm::vec3 u = glm::normalize(glm::cross(glm::vec3(0,1,0),n));
    glm::vec3 v = glm::cross(n,u);
    float fov = 45/(180.0 / PI);
    float d = (height/tan(fov/2))/2;
    glm::vec3 L = (cameraPosition-n*d) - u * (width/2) - v*(height/2);
    
    //generate primary rays
    Ray *cudarays;
    hipMallocManaged(&cudarays,totalRaysInSystem*sizeof(Ray));
    int blockSize = 256;
    int numBlocks = (totalRaysInSystem + blockSize -1)/blockSize;
    GeneratePrimaryRays<<<numBlocks,blockSize>>>(cudarays,totalRaysInSystem, L, u, v, cameraPosition);
    hipDeviceSynchronize();
    
    Ray *reflectedRays;
    hipMallocManaged(&reflectedRays,totalRaysInSystem*sizeof(Ray));
    
    Isect *cpuisectPoints = (Isect *)malloc(totalRaysInSystem*sizeof(Isect));;
    Isect *isectPoints;
    hipMallocManaged(&isectPoints,totalRaysInSystem*sizeof(Isect));
    int *nw,*ne,*sw,*se;
    hipMallocManaged(&nw,sizeof(int));
    hipMallocManaged(&ne,sizeof(int));
    hipMallocManaged(&sw,sizeof(int));
    hipMallocManaged(&se,sizeof(int));
    for(int i =0; i < 5; i++)
    {
        //Primary and secondary ray tracing
        RayIntersection<<<numBlocks,blockSize>>>(cudarays,totalRaysInSystem,reflectedRays,rootnode,isectPoints,nw,ne,sw,se);
        hipDeviceSynchronize();
        //shading intersection points
        Shade<<<numBlocks, blockSize>>>(isectPoints,totalRaysInSystem,scenelights,numlights,rootnode);
        hipDeviceSynchronize();
        
        hipMemcpy(cpuisectPoints, isectPoints, totalRaysInSystem*sizeof(Isect),hipMemcpyDeviceToHost);
    
        for(int i = 0; i<totalRaysInSystem;i++)
        {
            pixelcolorBuffer[359-cpuisectPoints[i].j][cpuisectPoints[i].i][0] += cpuisectPoints[i].color[0];
            pixelcolorBuffer[359-cpuisectPoints[i].j][cpuisectPoints[i].i][1] += cpuisectPoints[i].color[1];
            pixelcolorBuffer[359-cpuisectPoints[i].j][cpuisectPoints[i].i][2] += cpuisectPoints[i].color[2];
        }
        
        hipMemcpy(cudarays, reflectedRays, totalRaysInSystem*sizeof(Ray),hipMemcpyDeviceToDevice);
        
    }
    hipFree(cudarays);
    hipFree(reflectedRays);
    hipFree(isectPoints);
    
}

