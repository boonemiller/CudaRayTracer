#include "hip/hip_runtime.h"
//
//  bvh.cpp
//  RayTracer
//
//  Created by Bo Miller on 2/1/19.
//  Copyright © 2019 Bo Miller. All rights reserved.
//
#include "bvh.hpp"
#include <limits>
Node* constructTree(std::vector<SceneObject>& objects, Node* currentNode)
{
    if(objects.size() <= 3)
    {
        for(int i = 0; i<objects.size();i++)
        {
            currentNode->objs[i] = objects[i];
        }
        currentNode->numObjs = (int)objects.size();
        currentNode->isleaf = true;
        return currentNode;
    }
    
    Node* newLeftNode;
    hipMallocManaged(&newLeftNode,sizeof(Node));
    newLeftNode->left = NULL;
    newLeftNode->right = NULL;
    newLeftNode->isleaf = false;

    Node* newRightNode = (Node *) malloc(sizeof(Node));
    hipMallocManaged(&newRightNode,sizeof(Node));
    newRightNode->left = NULL;
    newRightNode->right = NULL;
    newRightNode->isleaf = false;
    
    std::vector<SceneObject> leftObjects;
    glm::vec3 midLeft;
    float maxLeftX = std::numeric_limits<float>::min();
    float minLeftX = std::numeric_limits<float>::max();
    float maxLeftY = std::numeric_limits<float>::min();
    float minLeftY = std::numeric_limits<float>::max();
    float maxLeftZ = std::numeric_limits<float>::min();
    float minLeftZ = std::numeric_limits<float>::max();
    glm::vec3 midRight;
    float maxRightX = std::numeric_limits<float>::min();
    float minRightX = std::numeric_limits<float>::max();
    float maxRightY = std::numeric_limits<float>::min();
    float minRightY = std::numeric_limits<float>::max();
    float maxRightZ = std::numeric_limits<float>::min();
    float minRightZ = std::numeric_limits<float>::max();
    std::vector<SceneObject> rightObjects;
    for(int i = 0; i < objects.size(); i++)
    {
        
        if(objects[i].position[currentNode->longestAxis] < currentNode->midpoint)
        {
            if(objects[i].position[0]-objects[i].radius < minLeftX)
                minLeftX = objects[i].position[0]-objects[i].radius;
            if(objects[i].position[1]-objects[i].radius < minLeftY)
                minLeftY = objects[i].position[1]-objects[i].radius;
            if(objects[i].position[2]-objects[i].radius < minLeftZ)
                minLeftZ = objects[i].position[2]-objects[i].radius;
            
            if(objects[i].position[0]+objects[i].radius > maxLeftX)
                maxLeftX = objects[i].position[0]+objects[i].radius;
            if(objects[i].position[1]+objects[i].radius > maxLeftY)
                maxLeftY = objects[i].position[1]+objects[i].radius;
            if(objects[i].position[2]+objects[i].radius > maxLeftZ)
                maxLeftZ = objects[i].position[2]+objects[i].radius;
            
            midLeft += objects[i].position;
            leftObjects.push_back(objects[i]);
        }
        else
        {
            if(objects[i].position[0]-objects[i].radius < minRightX)
                minRightX = objects[i].position[0]-objects[i].radius;
            if(objects[i].position[1]-objects[i].radius < minRightY)
                minRightY = objects[i].position[1]-objects[i].radius;
            if(objects[i].position[2]-objects[i].radius < minRightZ)
                minRightZ = objects[i].position[2]-objects[i].radius;
            
            if(objects[i].position[0]+objects[i].radius > maxRightX)
                maxRightX = objects[i].position[0]+objects[i].radius;
            if(objects[i].position[1]+objects[i].radius > maxRightY)
                maxRightY = objects[i].position[1]+objects[i].radius;
            if(objects[i].position[2]+objects[i].radius > maxRightZ)
                maxRightZ = objects[i].position[2]+objects[i].radius;
            midRight += objects[i].position;
            rightObjects.push_back(objects[i]);
        }
    }
    
    midLeft = glm::vec3(midLeft[0]/leftObjects.size(),midLeft[1]/leftObjects.size(),midLeft[2]/leftObjects.size());
    midRight = glm::vec3(midRight[0]/rightObjects.size(),midRight[1]/rightObjects.size(),midRight[2]/rightObjects.size());
    
    if(maxLeftX-minLeftX > maxLeftY - minLeftY)
    {
        if(maxLeftX-minLeftX > maxLeftZ - minLeftZ)
        {
            newLeftNode->longestAxis = 0;
            newLeftNode->midpoint = midLeft[0];
        }
    }
    if(maxLeftY-minLeftY > maxLeftX - minLeftX)
    {
        if(maxLeftY-minLeftY > maxLeftZ - minLeftZ)
        {
            newLeftNode->longestAxis = 1;
            newLeftNode->midpoint = midLeft[1];
        }
    }
    if(maxLeftZ - minLeftZ > maxLeftX - minLeftX)
    {
        if(maxLeftZ - minLeftZ > maxLeftY-minLeftY)
        {
            newLeftNode->longestAxis = 2;
            newLeftNode->midpoint = midLeft[2];
        }
    }
    
    if(maxRightX-minRightX > maxRightY - minRightY)
    {
        if(maxRightX-minRightX > maxRightZ - minRightZ)
        {
            newRightNode->longestAxis = 0;
            newRightNode->midpoint = midRight[0];
        }
    }
    if(maxRightY-minRightY > maxRightX - minRightX)
    {
        if(maxRightY-minRightY > maxRightZ - minRightZ)
        {
            newRightNode->longestAxis = 1;
            newRightNode->midpoint = midRight[1];
        }
    }
    if(maxRightZ-minRightZ > maxRightX - minRightX)
    {
        if(maxRightZ-minRightZ > maxRightY - minRightY)
        {
            newRightNode->longestAxis = 2;
            newRightNode->midpoint = midRight[2];
        }
    }
    newLeftNode->minX = minLeftX;
    newLeftNode->maxX = maxLeftX;
    newLeftNode->minY = minLeftY;
    newLeftNode->maxY = maxLeftY;
    newLeftNode->minZ = minLeftZ;
    newLeftNode->maxZ = maxLeftZ;
    
    newRightNode->minX = minRightX;
    newRightNode->maxX = maxRightX;
    newRightNode->minY = minRightY;
    newRightNode->maxY = maxRightY;
    newRightNode->minZ = minRightZ;
    newRightNode->maxZ = maxRightZ;
    
    currentNode->left  = constructTree(leftObjects, newLeftNode);
    currentNode->right = constructTree(rightObjects, newRightNode);
    
    return currentNode;
}



