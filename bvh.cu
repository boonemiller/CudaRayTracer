#include "hip/hip_runtime.h"
//
//  bvh.cpp
//  RayTracer
//
//  Created by Bo Miller on 2/1/19.
//  Copyright © 2019 Bo Miller. All rights reserved.
//
#include "bvh.hpp"
#include <limits>
#include <algorithm>
int nNum = 1;
Node* constructTree(std::vector<SceneObject *>& objects, Node*& currentNode, std::deque<Node *>& leafs, Node*& parentNode)
{
    if(objects.size() <= 64)
    {
        for(int i = 0; i<objects.size();i++)
        {
            currentNode->objs[i] = objects[i];
        }
        currentNode->numObjs = (int)objects.size();
        currentNode->isleaf = true;
	leafs.push_back(currentNode);       
	return currentNode;
    }
    
    Node* newLeftNode;
    hipMallocManaged(&newLeftNode,sizeof(Node));
    newLeftNode->parent = parentNode;
    newLeftNode->nodeNum = nNum++;
    newLeftNode->left = NULL;
    newLeftNode->right = NULL;
    newLeftNode->isleaf = false;

    Node* newRightNode;
    hipMallocManaged(&newRightNode,sizeof(Node));
    newRightNode->parent = parentNode;
    newRightNode->nodeNum = nNum++;
    newRightNode->left = NULL;
    newRightNode->right = NULL;
    newRightNode->isleaf = false;
    
    std::vector<SceneObject*> leftObjects;
    glm::vec3 midLeft;
    float maxLeftX = std::numeric_limits<float>::min();
    float minLeftX = std::numeric_limits<float>::max();
    float maxLeftY = std::numeric_limits<float>::min();
    float minLeftY = std::numeric_limits<float>::max();
    float maxLeftZ = std::numeric_limits<float>::min();
    float minLeftZ = std::numeric_limits<float>::max();
    glm::vec3 midRight;
    float maxRightX = std::numeric_limits<float>::min();
    float minRightX = std::numeric_limits<float>::max();
    float maxRightY = std::numeric_limits<float>::min();
    float minRightY = std::numeric_limits<float>::max();
    float maxRightZ = std::numeric_limits<float>::min();
    float minRightZ = std::numeric_limits<float>::max();
    std::vector<SceneObject*> rightObjects;
    for(int i = 0; i < objects.size(); i++)
    {
        float objmidpoint;
        if(objects[i]->sphere)
            objmidpoint = objects[i]->position[currentNode->longestAxis];
        else
        {
            objmidpoint = (objects[i]->v1[currentNode->longestAxis]+objects[i]->v2[currentNode->longestAxis]+objects[i]->v3[currentNode->longestAxis])/3.0f;
        }
        
        if(objmidpoint <= currentNode->midpoint)
        {
            if(objects[i]->sphere)
            {
                if(objects[i]->position[0]-objects[i]->radius < minLeftX)
                    minLeftX = objects[i]->position[0]-objects[i]->radius;
                if(objects[i]->position[1]-objects[i]->radius < minLeftY)
                    minLeftY = objects[i]->position[1]-objects[i]->radius;
                if(objects[i]->position[2]-objects[i]->radius < minLeftZ)
                    minLeftZ = objects[i]->position[2]-objects[i]->radius;
                
                if(objects[i]->position[0]+objects[i]->radius > maxLeftX)
                    maxLeftX = objects[i]->position[0]+objects[i]->radius;
                if(objects[i]->position[1]+objects[i]->radius > maxLeftY)
                    maxLeftY = objects[i]->position[1]+objects[i]->radius;
                if(objects[i]->position[2]+objects[i]->radius > maxLeftZ)
                    maxLeftZ = objects[i]->position[2]+objects[i]->radius;
                midLeft += objects[i]->position;
            }
            else if(objects[i]->triangle)
            {
                if(objects[i]->v1[0] < minLeftX)
                    minLeftX = objects[i]->v1[0];
                if(objects[i]->v1[1] < minLeftY)
                    minLeftY = objects[i]->v1[1];
                if(objects[i]->v1[2] < minLeftZ)
                    minLeftZ = objects[i]->v1[2];
                
                if(objects[i]->v1[0] > maxLeftX)
                    maxLeftX = objects[i]->v1[0];
                if(objects[i]->v1[1] > maxLeftY)
                    maxLeftY = objects[i]->v1[1];
                if(objects[i]->v1[2] > maxLeftZ)
                    maxLeftZ = objects[i]->v1[2];
                
                if(objects[i]->v2[0] < minLeftX)
                    minLeftX = objects[i]->v2[0];
                if(objects[i]->v2[1] < minLeftY)
                    minLeftY = objects[i]->v2[1];
                if(objects[i]->v2[2] < minLeftZ)
                    minLeftZ = objects[i]->v2[2];
                
                if(objects[i]->v2[0] > maxLeftX)
                    maxLeftX = objects[i]->v2[0];
                if(objects[i]->v2[1] > maxLeftY)
                    maxLeftY = objects[i]->v2[1];
                if(objects[i]->v2[2] > maxLeftZ)
                    maxLeftZ = objects[i]->v2[2];
                
                if(objects[i]->v3[0] < minLeftX)
                    minLeftX = objects[i]->v3[0];
                if(objects[i]->v3[1] < minLeftY)
                    minLeftY = objects[i]->v3[1];
                if(objects[i]->v3[2] < minLeftZ)
                    minLeftZ = objects[i]->v3[2];
                
                if(objects[i]->v3[0] > maxLeftX)
                    maxLeftX = objects[i]->v3[0];
                if(objects[i]->v3[1] > maxLeftY)
                    maxLeftY = objects[i]->v3[1];
                if(objects[i]->v3[2] > maxLeftZ)
                    maxLeftZ = objects[i]->v3[2];
                midLeft += glm::vec3((objects[i]->v1[0]+objects[i]->v2[0]+objects[i]->v3[0])/3.0f,(objects[i]->v1[1]+objects[i]->v2[1]+objects[i]->v3[1])/3.0f,(objects[i]->v1[2]+objects[i]->v2[2]+objects[i]->v3[2])/3.0f);
            }
            leftObjects.push_back(objects[i]);
        }
        else
        {
            if(objects[i]->sphere)
            {
                if(objects[i]->position[0]-objects[i]->radius < minRightX)
                    minRightX = objects[i]->position[0]-objects[i]->radius;
                if(objects[i]->position[1]-objects[i]->radius < minRightY)
                    minRightY = objects[i]->position[1]-objects[i]->radius;
                if(objects[i]->position[2]-objects[i]->radius < minRightZ)
                    minRightZ = objects[i]->position[2]-objects[i]->radius;
                
                if(objects[i]->position[0]+objects[i]->radius > maxRightX)
                    maxRightX = objects[i]->position[0]+objects[i]->radius;
                if(objects[i]->position[1]+objects[i]->radius > maxRightY)
                    maxRightY = objects[i]->position[1]+objects[i]->radius;
                if(objects[i]->position[2]+objects[i]->radius > maxRightZ)
                    maxRightZ = objects[i]->position[2]+objects[i]->radius;
                midRight += objects[i]->position;
            }
            else if(objects[i]->triangle)
            {
                if(objects[i]->v1[0] < minRightX)
                    minRightX = objects[i]->v1[0];
                if(objects[i]->v1[1] < minRightY)
                    minRightY = objects[i]->v1[1];
                if(objects[i]->v1[2] < minRightZ)
                    minRightZ = objects[i]->v1[2];
                
                if(objects[i]->v1[0] > maxRightX)
                    maxRightX = objects[i]->v1[0];
                if(objects[i]->v1[1] > maxRightY)
                    maxRightY = objects[i]->v1[1];
                if(objects[i]->v1[2] > maxRightZ)
                    maxRightZ = objects[i]->v1[2];
                
                if(objects[i]->v2[0] < minRightX)
                    minRightX = objects[i]->v2[0];
                if(objects[i]->v2[1] < minRightY)
                    minRightY = objects[i]->v2[1];
                if(objects[i]->v2[2] < minRightZ)
                    minRightZ = objects[i]->v2[2];
                
                if(objects[i]->v2[0] > maxRightX)
                    maxRightX = objects[i]->v2[0];
                if(objects[i]->v2[1] > maxRightY)
                    maxRightY = objects[i]->v2[1];
                if(objects[i]->v2[2] > maxRightZ)
                    maxRightZ = objects[i]->v2[2];
                
                if(objects[i]->v3[0] < minRightX)
                    minRightX = objects[i]->v3[0];
                if(objects[i]->v3[1] < minRightY)
                    minRightY = objects[i]->v3[1];
                if(objects[i]->v3[2] < minRightZ)
                    minRightZ = objects[i]->v3[2];
                
                if(objects[i]->v3[0] > maxRightX)
                    maxRightX = objects[i]->v3[0];
                if(objects[i]->v3[1] > maxRightY)
                    maxRightY = objects[i]->v3[1];
                if(objects[i]->v3[2] > maxRightZ)
                    maxRightZ = objects[i]->v3[2];
                midRight += glm::vec3((objects[i]->v1[0]+objects[i]->v2[0]+objects[i]->v3[0])/3.0f,(objects[i]->v1[1]+objects[i]->v2[1]+objects[i]->v3[1])/3.0f,(objects[i]->v1[2]+objects[i]->v2[2]+objects[i]->v3[2])/3.0f);
            }
            rightObjects.push_back(objects[i]);
        }
    }
    
    midLeft = glm::vec3(midLeft[0]/leftObjects.size(),midLeft[1]/leftObjects.size(),midLeft[2]/leftObjects.size());
    midRight = glm::vec3(midRight[0]/rightObjects.size(),midRight[1]/rightObjects.size(),midRight[2]/rightObjects.size());
    
    if(maxLeftX-minLeftX > maxLeftY - minLeftY)
    {
        if(maxLeftX-minLeftX > maxLeftZ - minLeftZ)
        {
            newLeftNode->longestAxis = 0;
            newLeftNode->midpoint = midLeft[0];
        }
    }
    if(maxLeftY-minLeftY > maxLeftX - minLeftX)
    {
        if(maxLeftY-minLeftY > maxLeftZ - minLeftZ)
        {
            newLeftNode->longestAxis = 1;
            newLeftNode->midpoint = midLeft[1];
        }
    }
    if(maxLeftZ - minLeftZ > maxLeftX - minLeftX)
    {
        if(maxLeftZ - minLeftZ > maxLeftY-minLeftY)
        {
            newLeftNode->longestAxis = 2;
            newLeftNode->midpoint = midLeft[2];
        }
    }
    
    if(maxRightX-minRightX > maxRightY - minRightY)
    {
        if(maxRightX-minRightX > maxRightZ - minRightZ)
        {
            newRightNode->longestAxis = 0;
            newRightNode->midpoint = midRight[0];
        }
    }
    if(maxRightY-minRightY > maxRightX - minRightX)
    {
        if(maxRightY-minRightY > maxRightZ - minRightZ)
        {
            newRightNode->longestAxis = 1;
            newRightNode->midpoint = midRight[1];
        }
    }
    if(maxRightZ-minRightZ > maxRightX - minRightX)
    {
        if(maxRightZ-minRightZ > maxRightY - minRightY)
        {
            newRightNode->longestAxis = 2;
            newRightNode->midpoint = midRight[2];
        }
    }
    newLeftNode->minX = minLeftX;
    newLeftNode->maxX = maxLeftX;
    newLeftNode->minY = minLeftY;
    newLeftNode->maxY = maxLeftY;
    newLeftNode->minZ = minLeftZ;
    newLeftNode->maxZ = maxLeftZ;
    
    newRightNode->minX = minRightX;
    newRightNode->maxX = maxRightX;
    newRightNode->minY = minRightY;
    newRightNode->maxY = maxRightY;
    newRightNode->minZ = minRightZ;
    newRightNode->maxZ = maxRightZ;

    currentNode->left  = constructTree(leftObjects, newLeftNode, leafs, newLeftNode);

    currentNode->right = constructTree(rightObjects, newRightNode, leafs, newRightNode);
    
    return currentNode;
}
void freeTree(Node*& root)
{
     
     if(root->isleaf)
	hipFree(root);
     else{
     	if(root->left != NULL)
	   freeTree(root->left);
	else if(root->right != NULL)
	   freeTree(root->right);
     }
     hipFree(root);
}

void refitTree(std::deque<Node *>& leafs)
{
   for(int i=0;i<leafs.size();i++)
	{
	    Node** parent = &leafs[i];
	    //(*parent)->parent->maxX = 12345;
	    while((*parent) != NULL)
	    {
		if((*parent)->isleaf)
	    	{
	    	    for(int o = 0; o<(*parent)->numObjs;o++)
            	    {
		        SceneObject* s = (*parent)->objs[o];
		        if(s->triangle)
		        {
		            if(s->v1[0] < (*parent)->minX)
                              (*parent)->minX = s->v1[0];
                    	    if(s->v1[1] < (*parent)->minY)
                      	      (*parent)->minY = s->v1[1];
                    	    if(s->v1[2] < (*parent)->minZ)
                       	      (*parent)->minZ = s->v1[2];
                
                    	    if(s->v1[0] > (*parent)->maxX)
                      	      (*parent)->maxX = s->v1[0];
                    	    if(s->v1[1] > (*parent)->maxY)
                              (*parent)->maxY = s->v1[1];
                    	    if(s->v1[2] > (*parent)->maxZ)
                      	      (*parent)->maxZ = s->v1[2];
                
                    	    if(s->v2[0] < (*parent)->minX)
                              (*parent)->minX = s->v2[0];
                    	    if(s->v2[1] < (*parent)->minY)
                              (*parent)->minY = s->v2[1];
                    	    if(s->v2[2] < (*parent)->minZ)
                      	      (*parent)->minZ = s->v2[2];
                
                    	    if(s->v2[0] > (*parent)->maxX)
                      	      (*parent)->maxX = s->v2[0];
                    	    if(s->v2[1] > (*parent)->maxY)
                              (*parent)->maxY = s->v2[1];
                    	    if(s->v2[2] > (*parent)->maxZ)
                              (*parent)->maxZ = s->v2[2];
                
                    	    if(s->v3[0] < (*parent)->minX)
                      	      (*parent)->minX = s->v3[0];
                    	    if(s->v3[1] < (*parent)->minY)
                      	      (*parent)->minY = s->v3[1];
                    	    if(s->v3[2] < (*parent)->minZ)
                      	      (*parent)->minZ = s->v3[2];
                
                    	    if(s->v3[0] > (*parent)->maxX)
                              (*parent)->maxX = s->v3[0];
                    	    if(s->v3[1] > (*parent)->maxY)
                      	      (*parent)->maxY = s->v3[1];
                    	    if(s->v3[2] > (*parent)->maxZ)
                     	      (*parent)->maxZ = s->v3[2];
		    	}
		    	if(s->sphere)
		    	{
			    if(s->position[0]-s->radius < (*parent)->minX)
                    	      (*parent)->minX = s->position[0]-s->radius;
                	    if(s->position[1]-s->radius < (*parent)->minY)
                    	      (*parent)->minY = s->position[1]-s->radius;
                	    if(s->position[2]-s->radius < (*parent)->minZ)
                    	      (*parent)->minZ = s->position[2]-s->radius;
                
                	    if(s->position[0]+s->radius > (*parent)->maxX)
                    	      (*parent)->maxX = s->position[0]+s->radius;
                	    if(s->position[1]+s->radius > (*parent)->maxY)
                    	      (*parent)->maxY = s->position[1]+s->radius;
                	    if(s->position[2]+s->radius > (*parent)->maxZ)
                    	      (*parent)->maxZ = s->position[2]+s->radius;
		    }
	        }
	    }
	    else
	    {
		if((*parent)->left->minX < (*parent)->minX)
		    (*parent)->minX = (*parent)->left->minX;
		if((*parent)->right->minX < (*parent)->minX)
		    (*parent)->minX = (*parent)->right->minX;

		if((*parent)->left->maxX > (*parent)->maxX)
		    (*parent)->maxX = (*parent)->left->maxX;
		if((*parent)->right->maxX > (*parent)->maxX)
		    (*parent)->maxX = (*parent)->right->maxX;

		if((*parent)->left->minY < (*parent)->minY)
		    (*parent)->minY = (*parent)->left->minY;
		if((*parent)->right->minY < (*parent)->minY)
		    (*parent)->minY = (*parent)->right->minY;

		if((*parent)->left->maxY > (*parent)->maxY)
		    (*parent)->maxY = (*parent)->left->maxY;
		if((*parent)->right->maxY > (*parent)->maxY)
		    (*parent)->maxY = (*parent)->right->maxY;

		if((*parent)->left->minZ < (*parent)->minZ)
		    (*parent)->minZ = (*parent)->left->minZ;
		if((*parent)->right->minZ < (*parent)->minZ)
		    (*parent)->minZ = (*parent)->right->minZ;

		if((*parent)->left->maxZ > (*parent)->maxZ)
		    (*parent)->maxZ = (*parent)->left->maxZ;

		if((*parent)->right->maxZ > (*parent)->maxZ)
		    (*parent)->maxZ = (*parent)->right->maxZ;

	    }
		parent = &(*parent)->parent;
	    }
	}
}



